#include <opencv2/videoio.hpp>
#include <opencv2/core.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/calib3d.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "obstacleKernel.h"

using namespace cv;
using namespace std;

void stereoObstacles(Mat* depth, Mat* obstacles, double maxDistance, int rows, int cols) {

    int BLOCK_SIZE = 16; // use square blocks
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Allocate device memory (GPU)
    uchar *d_depth, *d_obstacles;
    hipMalloc((void**) &d_obstacles, cols * rows * sizeof(uchar));
    hipMalloc((void**) &d_depth, cols * rows * sizeof(uchar));

    // Copy images from host (CPU RAM) to device (GPU)
    hipMemcpy(d_depth, depth->data, cols * rows * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_obstacles, obstacles->data,cols * rows * sizeof(uchar), hipMemcpyHostToDevice);

    // Launch kernel
    obstacleKernel<<<grid, block>>>(d_depth, d_obstalces, maxDistance, rows, cols);
    hipDeviceSynchronize();

    // Copy back the disparity map
    hipMemcpy(obstacles->data, d_obstacles, cols * rows * sizeof(uchar), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_left);
    hipFree(d_right);
    hipFree(d_depth);
}
