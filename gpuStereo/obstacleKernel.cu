#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <opencv2/core/cuda_types.hpp>
#include <opencv2/core.hpp>
#include "stereoObstacles.h"

// The actual CUDA kernel for obstacle detection
__global__ void obstacleKernel(const unsigned char* disparity,
                                      unsigned char* obstacles,
                                      double maxDistance,
                                      int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int halfWindow = 6;
    double minY = 100.0;

    //left camera parameters
    double baseline = 60.0;
    double fx = 543;
    double ox = 317.872;
    double fy = 721.241;
    double oy = 228.513;
    unsigned char pixel;
    double disparityChange = 0.0;

    //printf("max distance: %f\n", maxDistance);
    pixel = disparity[row * cols + col];

    double z = (double) pixel;

    double distance;

    /*
    if (z > 0 && z < maxDistance) {
        distance = baseline*fx/z;
    } else {
        distance = maxDistance + 1; // Mark as out of range
    }
    */
    if(z > 0) distance = baseline * fx / z;
    else distance = maxDistance + 1;
    


    if(distance > 0.0 && distance < maxDistance){
        //printf("distance: %f\n", distance);
        obstacles[row * cols + col] = (unsigned char)(255);
    } else {
        obstacles[row * cols + col] = (unsigned char)(0);
    }

/*
    double disp = (double)disparity[row*cols + col];

    //compute z the distance from camera
    double z = baseline*fx/disp;
    double z1 = baseline*fx/disp1;
    double z2 = baseline*fx/disp2;

    //compute x the side to side distnace. Neg to left
    double x = z * (ox - (double)col) / fx;

    // compute y the distance above the ground. y pos up
    double y = cameraHeight + z*((double)row - oy)/fy;
    double y1 = cameraHeight + z1*((double)(row - 1) - oy)/fy;
    double y2 = cameraHeight + z2*((double)(row + 1) - oy)/fy;

    //check if obstacle is within detection zone
    if(z > maxZ || fabs(x) > maxX){
        obstacles[row*cols+col] = 0;
        return;
    } 

    //

    int obstacleCol = (int)(cols*z/maxDistance);
    int obstacleRow = (int)(rows*(maxX+x)/(2*maxX));
*/
    
}