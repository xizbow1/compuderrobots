#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <opencv2/core/cuda_types.hpp>
#include <opencv2/core.hpp>
#include "stereoObstacles.h"

// The actual CUDA kernel for obstacle detection
__global__ void obstacleKernel(const unsigned char* disparity,
                                      unsigned char* obstacles,
                                      double maxDistance,
                                      int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int halfWindow = 6;

    //left camera parameters
    double baseline = 60.0;
    double fx = 543.0724862472363;
    double ox = 317.872;
    double fy = 721.241;
    double oy = 228.513;
    unsigned char pixel;
    double maxDisparity = 128.0;
    double disparityChange = 0.0;
    double cameraHeight = 150.0;
    double minY = 100.0;
    double maxY = 500.0;
    double maxX = 500.0;
    double maxZ = maxDistance;
    double minZ = baseline*fx/maxDisparity;
    

    //printf("max distance: %f\n", maxDistance);
    double distance;
    double disp = (double)disparity[row * cols + col];
    double z = 0.0;
    double x = z*(ox - (double)col)/fx;
    double y = cameraHeight + z*((double)row - oy)/fy;

    if(disp > 0.0){
        z = baseline * fx / disp;
        //printf("distance: %f\n", distance);
    } else {
        z = maxDistance + 1;
    }

    if(z > maxZ || z < minZ || fabs(x) > maxX || y < minY || y > maxY){
        obstacles[row * cols + col] = (unsigned char)(0);
        return;
    } else {
        obstacles[row * cols + col] = (unsigned char)(255);
    }
    /*
    if(distance > 0.0 && distance < maxDistance){
        printf("distance: %f\n", distance);
        obstacles[row * cols + col] = (unsigned char)(255);
    } else {
        obstacles[row * cols + col] = (unsigned char)(0);
    }
    */

/*
    double disp = (double)disparity[row*cols + col];

    //compute z the distance from camera
    double z = baseline*fx/disp;
    double z1 = baseline*fx/disp1;
    double z2 = baseline*fx/disp2;

    //compute x the side to side distnace. Neg to left
    double x = z * (ox - (double)col) / fx;

    // compute y the distance above the ground. y pos up
    double y = cameraHeight + z*((double)row - oy)/fy;
    double y1 = cameraHeight + z1*((double)(row - 1) - oy)/fy;
    double y2 = cameraHeight + z2*((double)(row + 1) - oy)/fy;

    //check if obstacle is within detection zone
    if(z > maxZ || fabs(x) > maxX){
        obstacles[row*cols+col] = 0;
        return;
    } 

    //

    int obstacleCol = (int)(cols*z/maxDistance);
    int obstacleRow = (int)(rows*(maxX+x)/(2*maxX));
*/
    
}