#include <opencv2/videoio.hpp>
#include <opencv2/core.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/calib3d.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "stereoKernel.h"

using namespace cv;
using namespace std;

void stereoDepth(Mat* left, Mat* right, Mat* depth, double maxDistance, int rows, int cols) {

    int BLOCK_SIZE = 16; // use square blocks
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Allocate device memory (GPU)
    uchar *d_left, *d_right, *d_depth;
    hipMalloc((void**) &d_left, cols * rows * sizeof(uchar));
    hipMalloc((void**) &d_right,cols * rows * sizeof(uchar));
    hipMalloc((void**) &d_depth,cols * rows * sizeof(uchar));

    // Copy images from host (CPU RAM) to device (GPU)
    hipMemcpy(d_left, left->data, cols * rows * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_right,right->data,cols * rows * sizeof(uchar), hipMemcpyHostToDevice);

    // Launch kernel
    stereoKernel<<<grid, block>>>(d_left, d_right, d_depth, maxDistance, rows, cols);
    hipDeviceSynchronize();

    // Copy back the disparity map
    hipMemcpy(depth->data, d_depth, cols * rows * sizeof(uchar), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_left);
    hipFree(d_right);
    hipFree(d_depth);
}
