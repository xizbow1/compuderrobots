#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

__global__ void stereoKernel(unsigned char* left, unsigned char* right, 
                            unsigned char* disparity, double maxDistance,
                            int rows, int cols){

    
// compute the row and col of the pixel to be processed
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;
int row1 = 124;
int col1 = 364;

    const int windowWidth = 13; //must be odd
    const int halfWindow = (windowWidth-1)/2;
    int disparityStep = 1
    int windowStep = 1;
    double minDisparity = 0.0;
    double maxDisparity = 64.0;
    double contrast;
    double contrastThreshold = 20;
    
    unsigned char leftPixel;
    unsigned char rightPixel;
    unsigned char centerPixel;
    double disp = 0.0;
    double distance;
    double sumSqDiff;
    double minSumSqDiff = (double)INT_MAX*(double)INT_MAX;
    double diff;
    double intensity, minIntensity, maxIntensity;


    if(row < halfWindow || row > rows-halfWindow || col < maxDisparity ||
        col < halfWindow || col > cols - halfWindow){
            disparity[row*cols+col] = 0;
            return;
        } 

    // Compute the contrast for left window
    // if contrast too low return
    minIntensity = (double)(left[row*cols+col]);
    maxIntensity = minIntensity;

    //printf("distance: %f\n", maxDistance);

    // Compute the sums within the windowsin each image
    for(int i = -halfWindow; i < halfWindow + 1; i++){
        for(int j = -halfWindow; j < halfWindow + 1; j++){
            intensity = (double)(left[(row + i) * cols + (col + j)]);
            if(intensity < minIntensity) minIntensity = intensity;
            if(intensity > maxIntensity) maxIntensity = intensity;
        }
    }

    // Ignore any contrast below the threshold
    contrast = maxIntensity - minIntensity;
    if(contrast < contrastThreshold){
        disparity[row*cols+col] = 0;
        return;
    }

    // Compute sum of squred differences each shifted window
    for(int k=0; k < maxDisparity; k++){
        sumSqDiff=0.0;
        for(int i = -halfWindow; i < halfWindow+1;i+=windowStep){
            for(int j = -halfWindow; j < halfWindow+1;j+=windowStep){
                    if(row + i < rows && col + j < cols && 0 <= col + j - k && col + j - k < cols){
                        leftPixel = left[(row+i)*cols+(col+j)];
                        rightPixel = right[(row+i)*cols+(col+j-k)];
                        diff = (double)leftPixel-(double)rightPixel;
                        sumSqDiff += fabs(diff);
                    }
            }
        }

        if(col == col1 && row == row1) printf("%f\n", sumSqDiff);
        //compute min sum square diff
        if(sumSqDiff < minSumSqDiff){
            minSumSqDiff = sumSqDiff;
            
            disp = (unsigned char)k;
        }

        
    }

    if(disp > 0){
        disparity[row * cols + col] = disp;
    }
    

    

/*
for (int k = 0; k < maxDisparity; k += disparityStep) {
    double sumLeft = 0.0, sumRight = 0.0, sumLeftSq = 0.0, sumRightSq = 0.0, sumProduct = 0.0;
    int count = 0;

    for (int i = -halfWindow; i <= halfWindow; i += windowStep) {
        for (int j = -halfWindow; j <= halfWindow; j += windowStep) {
            if (row + i >= 0 && row + i < rows &&
                col + j >= 0 && col + j < cols &&
                col + j - k >= 0 && col + j - k < cols) {
                leftPixel = left[(row + i) * cols + (col + j)];
                rightPixel = right[(row + i) * cols + (col + j - k)];

                sumLeft += leftPixel;
                sumRight += rightPixel;
                sumLeftSq += leftPixel * leftPixel;
                sumRightSq += rightPixel * rightPixel;
                sumProduct += leftPixel * rightPixel;
                count++;
            }
        }
    }

    if (count > 0) {
        double meanLeft = sumLeft / count;
        double meanRight = sumRight / count;
        double numerator = sumProduct - count * meanLeft * meanRight;
        double denominator = sqrt((sumLeftSq - count * meanLeft * meanLeft) *
                                  (sumRightSq - count * meanRight * meanRight));

        double ncc = (denominator > 1e-6) ? numerator / denominator : 0.0;

        if (ncc > minSumSqDiff) { // Maximize NCC
            minSumSqDiff = ncc;
            disp = k;
        }
    }
}
disparity[row * cols + col] = (unsigned char)(disp);
*/
}
