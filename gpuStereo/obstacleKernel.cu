#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <opencv2/core/cuda_types.hpp>
#include <opencv2/core.hpp>
#include "stereoObstacles.h"

using namespace cv;
using namespace cv::cuda;

// The actual CUDA kernel for obstacle detection
__global__ void obstacleKernel(const unsigned char* disparity,
                                      unsigned char* obstacles,
                                      double maxDistance,
                                      int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    //left camera parameters
    double baseline = 60.0;
    double fx = 578.0;
    double ox;
    double fy;
    double oy;
    unsigned char pixel;
    double disparityChange = 0.0;

    pixel = disparity[rows * cols + col];

    double z = (double) pixel;

    double distance;
    if(z > 0) distance = (baseline * fx) / z;
    else distance = 0.0;

    if(distance > 0.0 && distance < maxDistance){
        obstacles[row * cols + col] = (unsigned char)(255);
    } else {
        obstacles[row * cols + col] = (unsigned char) 0;
    }
/*
        //obstacle map resolution mm
    double maxX = 2000;

    double disp = (double)disparity[row*col + col];

    //compute z the distance from camera
    double z = baseline*fx/disp;

    //compute x the side to side distnace. Neg to left
    double x = z * (col - ox) / fx;

    //compute obstacle location on the map
    int obstacleCol, obstacleRow;
    if(z>maxDistance || fabs(x)>maxX) return;

    obstacleCol = (int)(cols*z/maxDistance);
    obstacleRow = (int)(rows*(maxX+x)/(2*maxX));

*/


}