#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

__global__ void stereoKernel(unsigned char* left, unsigned char* right, unsigned char* depth, double maxDistance, int rows, int cols){

    
// compute the row and col of the pixel to be processed
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;

// put your stereo matching code here
// This code should only be for one pixel
// See the video I posted on acceleration stereo on the GPU 


    const int windowWidth = 11; //must be odd
    const int halfWindow = (windowWidth-1)/2;
    double baseLine = 60.0;
    double focalLength = 578.0;
    double maxDisparity = 71;

    unsigned char leftPixel;
    unsigned char rightPixel;
    int disparity;
    double distance;
    double sumSqDiff;
    int minSumSqDiff = INT_MAX;
    int diff;

    if(row < halfWindow || row > rows-halfWindow || col < halfWindow || col > cols - halfWindow) return;

    //compute sum of squred differecnes each shifted window

    for(int k=0; k<maxDisparity;k++){
        sumSqDiff=0.0;
        for(int i = -halfWindow; i<halfWindow+1;i++){
            for(int j = -halfWindow; j<halfWindow+1;j++){

                    leftPixel = left[(row+i)*cols+(col+j)];
                    rightPixel = right[(row+i)*cols+(col+j-k)];
                    diff = leftPixel-rightPixel;
                    sumSqDiff += diff*diff;
            }
        }

        
        //compute min sum square diff
        if(sumSqDiff < minSumSqDiff){
            minSumSqDiff = sumSqDiff;
            disparity = k;
        }
    }

    //if we have a valid disparity, compute the distance and save it
    if(disparity > 0){
        distance = baseLine*focalLength/disparity;
        depth[row*cols+col] = (unsigned char) (255.0*distance/maxDistance);
    }else {
        depth[row*cols+col]=255;
    }
}
