#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

__global__ void stereoKernel(unsigned char* left, unsigned char* right, unsigned char* depth,
                             double maxDistance, int rows, int cols){

// compute the row and col of the pixel to be processed
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;

// put your stereo matching code here
// This code should only be for one pixel
// See the video I posted on acceleration stereo on the GPU



}
