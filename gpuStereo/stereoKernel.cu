#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

__global__ void stereoKernel(unsigned char* left, unsigned char* right, 
                            unsigned char* disparity, double maxDistance,
                            int rows, int cols){

    
// compute the row and col of the pixel to be processed
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;

    const int windowWidth = 5; //must be odd
    const int halfWindow = (windowWidth-1)/2;
    int disparityStep = 2;
    int windowStep = 2;
    double maxDisparity = 56.5;
    double contrast;
    double contrastThreshold = 5;
    
    unsigned char leftPixel;
    unsigned char rightPixel;
    unsigned char centerPixel;
    double disp = 0.0;
    double distance;
    double sumSqDiff;
    double minSumSqDiff = (double)INT_MAX*(double)INT_MAX;
    double diff;
    double intensity, minIntensity, maxIntensity;


    if(row < halfWindow || row > rows-halfWindow ||
        col < halfWindow || col > cols - halfWindow) return;

    // Compute the contrast for left window
    // if contrast too low return
    minIntensity = (double)(left[row*cols+col]);
    maxIntensity = minIntensity;

    // Compute the sums within the windowsin each image
    for(int i = -halfWindow; i < halfWindow + 1; i++){
        for(int j = -halfWindow; j < halfWindow + 1; j++){
            intensity = (double)(left[(row + i) * cols + (col + j)]);
            if(intensity < minIntensity) minIntensity = intensity;
            if(intensity > maxIntensity) maxIntensity = intensity;
        }
    }

    // Ignore any contrast below the threshold
    contrast = maxIntensity - minIntensity;
    if(contrast < contrastThreshold) return;

    // Compute sum of squred differences each shifted window
    for(int k=0; k < maxDisparity; k++){
        sumSqDiff=0.0;
        for(int i = -halfWindow; i < halfWindow+1;i++){
            for(int j = -halfWindow; j < halfWindow+1;j++){
                    if(row + i < rows && col + j < cols && 0 <= col + j - k && col + j - k < cols){
                        leftPixel = left[(row+i)*cols+(col+j)];
                        rightPixel = right[(row+i)*cols+(col+j-k)];
                        diff = (double)leftPixel-(double)rightPixel;
                        sumSqDiff += fabs(diff);
                    }
            }
        }

        //compute min sum square diff
        if(sumSqDiff < minSumSqDiff){
            minSumSqDiff = sumSqDiff;
            disp = (unsigned char)k;
        }
    }

    disparity[row*cols+col] = disp;

    /*
    // Replace SSD with NCC for better matching
for (int k = 0; k < maxDistance
; k += disparityStep) {
    double sumLeft = 0.0, sumRight = 0.0, sumLeftSq = 0.0, sumRightSq = 0.0, sumProduct = 0.0;
    int count = 0;

    for (int i = -halfWindow; i < halfWindow + 1; i += windowStep) {
        for (int j = -halfWindow; j < halfWindow + 1; j += windowStep) {
            if (row + i < rows && col + j < cols && 0 <= col + j - k && col + j - k < cols) {
                leftPixel = left[(row + i) * cols + (col + j)];
                rightPixel = right[(row + i) * cols + (col + j - k)];

                sumLeft += leftPixel;
                sumRight += rightPixel;
                sumLeftSq += leftPixel * leftPixel;
                sumRightSq += rightPixel * rightPixel;
                sumProduct += leftPixel * rightPixel;
                count++;
            }
        }
    }

    if (count > 0) {
        double meanLeft = sumLeft / count;
        double meanRight = sumRight / count;
        double numerator = sumProduct - count * meanLeft * meanRight;
        double denominator = sqrt((sumLeftSq - count * meanLeft * meanLeft) * 
                                  (sumRightSq - count * meanRight * meanRight));

        double ncc = (denominator > 0) ? numerator / denominator : 0.0;

        if (ncc > minSumSqDiff) { // Maximize NCC instead of minimizing SSD
            minSumSqDiff = ncc;
            disp = k;
        } 
    }
        disparity[row*cols+col] = (unsigned char) (disp);
}
    */

}
